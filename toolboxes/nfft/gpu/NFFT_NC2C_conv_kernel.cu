#include "hip/hip_runtime.h"
/*
  CUDA implementation of the NFFT.

  -----------

  Accelerating the Non-equispaced Fast Fourier Transform on Commodity Graphics Hardware.
  T.S. Sørensen, T. Schaeffter, K.Ø. Noe, M.S. Hansen. 
  IEEE Transactions on Medical Imaging 2008; 27(4):538-547.

  Real-time Reconstruction of Sensitivity Encoded Radial Magnetic Resonance Imaging Using a Graphics Processing Unit.
  T.S. Sørensen, D. Atkinson, T. Schaeffter, M.S. Hansen.
  IEEE Transactions on Medical Imaging 2009; 28(12): 1974-1985. 
*/

//
// There is no header file accompanying this kernel, so it makes most sense to read the code/file from the end and upwards
//

//
// Transfer result from shared memory to global memory.
//

template<class REAL> __inline__ __device__ void 
NFFT_H_output( unsigned int number_of_batches, complext<REAL>* __restrict__ image,
	       unsigned int double_warp_size_power, unsigned int number_of_domains, 
	       unsigned int globalThreadId, unsigned int sharedMemFirstCellIdx )
{

  REAL *shared_mem = (REAL*) _shared_mem;
  
  for( unsigned int batch=0; batch<number_of_batches; batch++ ){
    complext<REAL>cell_coefficient;
    cell_coefficient._real = shared_mem[sharedMemFirstCellIdx+(batch<<double_warp_size_power)];
    cell_coefficient._imag = shared_mem[sharedMemFirstCellIdx+(batch<<double_warp_size_power)+warpSize];
    image[(batch*gridDim.y+blockIdx.y)*number_of_domains+globalThreadId] = cell_coefficient;
  }
}


template<class REAL, unsigned int D> __inline__ __device__ void
NFFT_H_convolve( typename reald<REAL,D>::Type alpha, typename reald<REAL,D>::Type beta, REAL W, 
		 unsigned int number_of_samples, unsigned int number_of_batches, unsigned int number_of_domains,
		 const vector_td<REAL,D> * __restrict__ traj_positions, complext<REAL>*samples, const unsigned int * __restrict__ tuples_last,
		 const unsigned int * __restrict__ bucket_begin, const unsigned int * __restrict__ bucket_end,
		 unsigned int double_warp_size_power, REAL half_W, REAL one_over_W, vector_td<REAL,D> matrix_size_os_real, 
		 unsigned int globalThreadId, vector_td<unsigned int,D> domainPos, unsigned int sharedMemFirstCellIdx, bool sqrt_kernel )
{

  REAL *shared_mem = (REAL*) _shared_mem;

  // Cell position as reald
  vector_td<REAL,D> cell_pos = vector_td<REAL,D>( domainPos );
  
  // Convolve samples onto the domain (shared memory)
  const unsigned int frame_offset = blockIdx.y*number_of_domains;
  for( unsigned int i=bucket_begin[globalThreadId+frame_offset]; i<bucket_end[globalThreadId+frame_offset]; i++ )
    {
      // Safety precaution TODO
      unsigned int sampleIdx = tuples_last[i];

      // Safety precaution TODO
      vector_td<REAL,D> sample_pos = traj_positions[sampleIdx];
      
      // Calculate the distance between the cell and the sample
      vector_td<REAL,D> delta = abs(sample_pos-cell_pos);
      vector_td<REAL,D> half_W_vec( half_W );
  
      // Check if sample will contribute
      if( weak_greater(delta, half_W_vec ))
	continue;
      
      // Compute convolution weights
      float weight = KaiserBessel<REAL>( delta, matrix_size_os_real, one_over_W, beta );
      if(sqrt_kernel){ weight = sqrt(weight); }
      
      // Safety measure
      if( !isfinite(weight) )
      	continue;
      
      // Apply Kaiser-Bessel filter to input images
      for( unsigned int batch=0; batch<number_of_batches; batch++ ){
	
	complext<REAL>sample_val = samples[sampleIdx+batch*gridDim.y*number_of_samples];

	// Apply filter to shared memory domain. 
	shared_mem[sharedMemFirstCellIdx+(batch<<double_warp_size_power)] += (weight*sample_val._real);
	shared_mem[sharedMemFirstCellIdx+(batch<<double_warp_size_power)+warpSize] += (weight*sample_val._imag);
      }
    }
}

//
// kernel main
//

template<class REAL, unsigned int D> __global__ void
NFFT_H_convolve_kernel( typename reald<REAL,D>::Type alpha, typename reald<REAL,D>::Type beta, REAL W,
			vector_td<unsigned int,D> domain_count_grid, unsigned int number_of_samples, unsigned int number_of_batches,
			const vector_td<REAL,D> * __restrict__ traj_positions, complext<REAL>* __restrict__ image, complext<REAL>* __restrict__ samples,
			const unsigned int * __restrict__ tuples_last, const unsigned int * __restrict__ bucket_begin, const unsigned int * __restrict__ bucket_end,
			unsigned int double_warp_size_power,
			REAL half_W, REAL one_over_W, vector_td<REAL,D> matrix_size_os_real, bool sqrt_kernel = false )
{
  
  // Global thread index
  const unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

  // Number of domains
  const unsigned int number_of_domains = prod(domain_count_grid);

  // Check if we are within bounds
  if( index >= number_of_domains )
    return;
  
  // Mapped global thread index (actually we don't use a map currently)
  const unsigned int domainIdx = index; 

  // Compute global domain position
  const vector_td<unsigned int,D> domainPos = idx_to_co<D>( domainIdx, domain_count_grid );
	
  // Number of cells
  const unsigned int num_reals = number_of_batches<<1;

  // All shared memory floats corresponding to domain 'threadIdx.x' is located in bank threadIdx.x%warp_size to limit bank conflicts
  const unsigned int scatterSharedMemStart = (threadIdx.x/warpSize)*warpSize;
  const unsigned int scatterSharedMemStartOffset = threadIdx.x&(warpSize-1); // a faster way of saying (threadIdx.x%warpSize) 
  const unsigned int sharedMemFirstCellIdx = scatterSharedMemStart*num_reals + scatterSharedMemStartOffset;

  REAL *shared_mem = (REAL*) _shared_mem;

  // Initialize shared memory
  for( unsigned int i=0; i<num_reals; i++ )
    shared_mem[sharedMemFirstCellIdx+warpSize*i] = REAL(0);
  
  // Compute NFFT using arbitrary sample trajectories.
  NFFT_H_convolve<REAL, D>
    ( alpha, beta, W, number_of_samples, number_of_batches, number_of_domains,
      traj_positions, samples, tuples_last, bucket_begin, bucket_end,
      double_warp_size_power, half_W, one_over_W,  matrix_size_os_real, index, domainPos, sharedMemFirstCellIdx, sqrt_kernel );
  
  // Output k-space image to global memory
  NFFT_H_output<REAL>( number_of_batches, image, double_warp_size_power, number_of_domains, index, sharedMemFirstCellIdx );
}
