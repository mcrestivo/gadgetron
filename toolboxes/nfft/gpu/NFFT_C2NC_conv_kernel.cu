#include "hip/hip_runtime.h"
/*
  CUDA implementation of the NFFT.

  -----------

  Accelerating the Non-equispaced Fast Fourier Transform on Commodity Graphics Hardware.
  T.S. Sørensen, T. Schaeffter, K.Ø. Noe, M.S. Hansen. 
  IEEE Transactions on Medical Imaging 2008; 27(4):538-547.

  Real-time Reconstruction of Sensitivity Encoded Radial Magnetic Resonance Imaging Using a Graphics Processing Unit.
  T.S. Sørensen, D. Atkinson, T. Schaeffter, M.S. Hansen.
  IEEE Transactions on Medical Imaging 2009; 28(12): 1974-1985. 
*/

//
// There is no header file accompanying this kernel, so it makes most sense to read the code/file from the end and upwards
//

//
// Transfer result from shared memory to global memory.
//

template<class REAL> __inline__ __device__ void 
NFFT_output( unsigned int number_of_samples, unsigned int number_of_batches, complext<REAL> * __restrict__ samples,
	     unsigned int double_warp_size_power, unsigned int globalThreadId, unsigned int sharedMemFirstSampleIdx, bool accumulate )
{
  
  REAL *shared_mem = (REAL*) _shared_mem;
  
  for( unsigned int batch=0; batch<number_of_batches; batch++ ){
    complext<REAL>sample_value;
    sample_value._real = shared_mem[sharedMemFirstSampleIdx+(batch<<double_warp_size_power)];
    sample_value._imag = shared_mem[sharedMemFirstSampleIdx+(batch<<double_warp_size_power)+warpSize];

    unsigned int out_idx = (batch*gridDim.y+blockIdx.y)*number_of_samples + globalThreadId;

    if( accumulate ) sample_value += samples[out_idx];
    samples[out_idx] = sample_value;
  }
}

template<unsigned int D> __inline__ __device__ static void
resolve_wrap( vector_td<int,D> &grid_position, vector_td<unsigned int,D> &matrix_size_os )
{
  vector_td<int,D> zero(0);
  grid_position += vector_less(grid_position, zero)*matrix_size_os;
  grid_position -= vector_greater_equal(grid_position, matrix_size_os)* matrix_size_os;
}

template<class REAL, unsigned int D> __inline__ __device__ void
NFFT_iterate_body( typename reald<REAL,D>::Type alpha, typename reald<REAL,D>::Type beta, REAL W, 
		   vector_td<unsigned int, D> matrix_size_os, unsigned int number_of_batches, complext<REAL> * __restrict__ image,
		   unsigned int double_warp_size_power, REAL half_W, REAL one_over_W, vector_td<REAL,D> matrix_size_os_real, unsigned int sharedMemFirstSampleIdx,
		   vector_td<REAL,D> sample_position, vector_td<int,D> grid_position, bool sqrt_kernel )
{
      
  // Calculate the distance between current sample and the grid cell
  vector_td<REAL,D> grid_position_real = vector_td<REAL,D>(grid_position);
  const vector_td<REAL,D> delta = abs(sample_position-grid_position_real);
  const vector_td<REAL,D> half_W_vec(half_W );
  
  // If cell too distant from sample then move on to the next cell
  if( weak_greater( delta, half_W_vec ))
    return;

  // Compute convolution weight.
  
  REAL weight =  KaiserBessel<REAL>( delta, matrix_size_os_real, one_over_W, beta );
  if( sqrt_kernel){ weight = sqrt(weight); }

  // Safety measure. We have occationally observed a NaN from the KaiserBessel computation
  if( !isfinite(weight) )
    return;

  // Resolve wrapping of grid position
  resolve_wrap<D>( grid_position, matrix_size_os);

  REAL *shared_mem = (REAL*) _shared_mem;
  
  for( unsigned int batch=0; batch<number_of_batches; batch++ ){
    
    // Read the grid cell value from global memory
    const complext<REAL> grid_value = 
      image[ (batch*gridDim.y+blockIdx.y)*prod(matrix_size_os) + co_to_idx<D>( vector_td<unsigned int, D>(grid_position), matrix_size_os ) ];
    
    // Add 'weight*grid_value' to the samples in shared memory
    shared_mem[sharedMemFirstSampleIdx+(batch<<double_warp_size_power)] += (weight*grid_value._real);
    shared_mem[sharedMemFirstSampleIdx+(batch<<double_warp_size_power)+warpSize] += (weight*grid_value._imag);
  }
}

//
// This method is deliberately overloaded in 'UINTd' (rather than templetized) to improve performance of the loop iteration
//

template<class REAL> __inline__ __device__ void
NFFT_iterate( typename reald<REAL,1>::Type alpha, typename reald<REAL,1>::Type beta, REAL W, 
	      vector_td<unsigned int,1> matrix_size_os, unsigned int number_of_batches, complext<REAL> * __restrict__ image,
	      unsigned int double_warp_size_power, REAL half_W, REAL one_over_W, vector_td<REAL,1> matrix_size_os_real, unsigned int sharedMemFirstSampleIdx,
	      vector_td<REAL,1> sample_position, vector_td<int,1> lower_limit, vector_td<int,1> upper_limit, bool sqrt_kernel )
{
  // Iterate through all grid cells influencing the corresponding sample
  for( int x = lower_limit.vec[0]; x<=upper_limit.vec[0]; x++ ){
    
    const intd<1>::Type grid_position(x);
    
    NFFT_iterate_body<REAL,1>( alpha, beta, W, matrix_size_os, number_of_batches, image, double_warp_size_power, half_W, 
			       one_over_W, matrix_size_os_real, sharedMemFirstSampleIdx, sample_position, grid_position, sqrt_kernel );
  }
}

//
// This method is deliberately overloaded in 'UINTd' (rather than templetized) to improve performance of the loop iteration
//

template<class REAL> __inline__ __device__ void
NFFT_iterate( typename reald<REAL,2>::Type alpha, typename reald<REAL,2>::Type beta, REAL W, 
	      vector_td<unsigned int,2> matrix_size_os, unsigned int number_of_batches, complext<REAL> * __restrict__ image,
	      unsigned int double_warp_size_power, REAL half_W, REAL one_over_W, vector_td<REAL,2> matrix_size_os_real, unsigned int sharedMemFirstSampleIdx,
	      vector_td<REAL,2> sample_position, vector_td<int,2> lower_limit, vector_td<int,2> upper_limit, bool sqrt_kernel )
{
  // Iterate through all grid cells influencing the corresponding sample
  for( int y = lower_limit.vec[1]; y<=upper_limit.vec[1]; y++ ){
    for( int x = lower_limit.vec[0]; x<=upper_limit.vec[0]; x++ ){
      
      const intd<2>::Type grid_position(x,y);
      
      NFFT_iterate_body<REAL,2>( alpha, beta, W, matrix_size_os, number_of_batches, image, double_warp_size_power, half_W, 
				 one_over_W, matrix_size_os_real, sharedMemFirstSampleIdx, sample_position, grid_position, sqrt_kernel );
    }
  }
}

//
// This method is deliberately overloaded in 'd' (rather than templetized) to improve performance of the loop iteration
//

template<class REAL> __inline__ __device__ void
NFFT_iterate( typename reald<REAL,3>::Type alpha, typename reald<REAL,3>::Type beta, REAL W, 
	      vector_td<unsigned int,3> matrix_size_os, unsigned int number_of_batches, complext<REAL> * __restrict__ image,
	      unsigned int double_warp_size_power, REAL half_W, REAL one_over_W, vector_td<REAL,3> matrix_size_os_real, unsigned int sharedMemFirstSampleIdx,
	      vector_td<REAL,3> sample_position, vector_td<int,3> lower_limit, vector_td<int,3> upper_limit, bool sqrt_kernel )
{
  // Iterate through all grid cells influencing the corresponding sample
  for( int z = lower_limit.vec[2]; z<=upper_limit.vec[2]; z++ ){
    for( int y = lower_limit.vec[1]; y<=upper_limit.vec[1]; y++ ){
      for( int x = lower_limit.vec[0]; x<=upper_limit.vec[0]; x++ ){
	
	const intd<3>::Type grid_position(x,y,z);
	
	NFFT_iterate_body<REAL,3>( alpha, beta, W, matrix_size_os, number_of_batches, image, double_warp_size_power, half_W, 
				   one_over_W, matrix_size_os_real, sharedMemFirstSampleIdx, sample_position, grid_position, sqrt_kernel );
      }
    }
  }
}

//
// This method is deliberately overloaded in 'd' (rather than templetized) to improve performance of the loop iteration
//

template<class REAL> __inline__ __device__ void
NFFT_iterate( typename reald<REAL,4>::Type alpha, typename reald<REAL,4>::Type beta, REAL W, 
	      vector_td<unsigned int,4> matrix_size_os, unsigned int number_of_batches, complext<REAL> * __restrict__ image,
	      unsigned int double_warp_size_power, REAL half_W, REAL one_over_W, vector_td<REAL,4> matrix_size_os_real, unsigned int sharedMemFirstSampleIdx,
	      vector_td<REAL,4> sample_position, vector_td<int,4> lower_limit, vector_td<int,4> upper_limit, bool sqrt_kernel )
{
  // Iterate through all grid cells influencing the corresponding sample
  for( int w = lower_limit.vec[3]; w<=upper_limit.vec[3]; w++ ){
    for( int z = lower_limit.vec[2]; z<=upper_limit.vec[2]; z++ ){
      for( int y = lower_limit.vec[1]; y<=upper_limit.vec[1]; y++ ){
	for( int x = lower_limit.vec[0]; x<=upper_limit.vec[0]; x++ ){
	  
	  const intd<4>::Type grid_position(x,y,z,w);
	  
	  NFFT_iterate_body<REAL,4>( alpha, beta, W, matrix_size_os, number_of_batches, image, double_warp_size_power, half_W, 
				     one_over_W, matrix_size_os_real, sharedMemFirstSampleIdx, sample_position, grid_position, sqrt_kernel );
	}
      }
    }
  }
}

template<class REAL, unsigned int D> __inline__ __device__ void
NFFT_convolve( typename reald<REAL,D>::Type alpha, typename reald<REAL,D>::Type beta, REAL W, 
	       vector_td<unsigned int, D> matrix_size_os, vector_td<unsigned int, D> matrix_size_wrap, 
	       unsigned int number_of_samples, unsigned int number_of_batches, const vector_td<REAL,D> * __restrict__ traj_positions, complext<REAL> * __restrict__ image,
	       unsigned int double_warp_size_power, REAL half_W, REAL one_over_W, vector_td<REAL,D> matrix_size_os_real,
	       unsigned int globalThreadId, unsigned int sharedMemFirstSampleIdx, bool sqrt_kernel )
{
  
  // Sample position to convolve onto
  // Computed in preprocessing, which included a wrap zone. Remove this wrapping.
  const vector_td<REAL,D> half_wrap_real = vector_td<REAL,D>(matrix_size_wrap>>1);
  const vector_td<REAL,D> sample_position = traj_positions[globalThreadId+blockIdx.y*number_of_samples]-half_wrap_real;
  
  // Half the kernel width
  const vector_td<REAL,D> half_W_vec( half_W );
  
  // Limits of the subgrid to consider
  const vector_td<int,D> lower_limit = vector_td<int,D>( ceil(sample_position-half_W_vec));
  const vector_td<int,D> upper_limit = vector_td<int,D>( floor(sample_position+half_W_vec));

  // Accumulate contributions from the grid
  NFFT_iterate<REAL>( alpha, beta, W, matrix_size_os, number_of_batches, image, double_warp_size_power, 
		      half_W, one_over_W, matrix_size_os_real, sharedMemFirstSampleIdx, sample_position, lower_limit, upper_limit, sqrt_kernel );
}

//
// kernel main
//

template<class REAL, unsigned int D> __global__ void
NFFT_convolve_kernel( typename reald<REAL,D>::Type alpha, typename reald<REAL,D>::Type beta, REAL W, 
		      vector_td<unsigned int, D> matrix_size_os, vector_td<unsigned int, D> matrix_size_wrap,
		      unsigned int number_of_samples, unsigned int number_of_batches, 
		      const vector_td<REAL,D> * __restrict__ traj_positions, complext<REAL> *image,  complext<REAL> * __restrict__ samples,
		      unsigned int double_warp_size_power, REAL half_W, REAL one_over_W, bool accumulate, vector_td<REAL,D> matrix_size_os_real, bool sqrt_kernel = false )
{

  // Global thread number	
  const unsigned int globalThreadId = (blockIdx.x*blockDim.x+threadIdx.x);

  // Check if we are within bounds
  if( globalThreadId >= number_of_samples )
    return;
  
  // Number of reals to compute/output per thread
  const unsigned int num_reals = number_of_batches<<1;
  
  // All shared memory reals corresponding to domain 'threadIdx.x' are located in bank threadIdx.x%warp_size to limit bank conflicts
  const unsigned int scatterSharedMemStart = (threadIdx.x/warpSize)*warpSize;
  const unsigned int scatterSharedMemStartOffset = threadIdx.x&(warpSize-1); // a faster way of saying (threadIdx.x%warpSize) 
  const unsigned int sharedMemFirstSampleIdx = scatterSharedMemStart*num_reals + scatterSharedMemStartOffset;

  REAL *shared_mem = (REAL*) _shared_mem;
  const REAL zero = REAL(0);

  // Initialize shared memory
  for( unsigned int i=0; i<num_reals; i++ )
    shared_mem[sharedMemFirstSampleIdx+warpSize*i] = zero;
  
  // Compute NFFT using arbitrary sample trajectories
  NFFT_convolve<REAL,D>( alpha, beta, W, matrix_size_os, matrix_size_wrap, number_of_samples, number_of_batches, 
			 traj_positions, image, double_warp_size_power, half_W, one_over_W, 
			 matrix_size_os_real, globalThreadId, sharedMemFirstSampleIdx, sqrt_kernel );
  
  // Output k-space image to global memory
  NFFT_output<REAL>( number_of_samples, number_of_batches, samples, double_warp_size_power, globalThreadId, sharedMemFirstSampleIdx, accumulate );
}

